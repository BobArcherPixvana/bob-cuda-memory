#include "hip/hip_runtime.h"

#include "hip/hiprtc.h"

#include <algorithm>
#include <chrono>
#include <cstddef>
#include <cstdint>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

void deviceQuery();

const char* logFile("log.txt");

#define LOG(exp)                                                                                   \
    do                                                                                             \
    {                                                                                              \
        std::ofstream ostr(logFile, std::ios::app);                                                \
        ostr << exp;                                                                               \
        std::cout << exp;                                                                          \
    } while(false)

#define INSPECT(exp) LOG(#exp << ": " << (exp) << "\n")
#define WHERE LOG(__FILE__ << ", " << __LINE__ << "\n")

namespace
{
std::string format(size_t i)
{
    std::ostringstream ostr;
    ostr << i;

    std::string s(ostr.str().c_str());
    std::string result;

    for(int i = 0; i < int(s.size()); ++i)
    {
        int const j(int(s.size()) - i - 1);

        if(i != 0 && i % 3 == 0)
        {
            result += ",";
        }

        result += s[j];
    }

    std::reverse(result.begin(), result.end());

    return result;
}

void writeResult(hipError_t result, std::string const& description)
{
    std::cout << std::setw(25) << std::left << description << " : ";
    std::cout << result << " ";
    std::cout << hipGetErrorName(result) << " ";
    std::cout << hipGetErrorString(result) << "\n";

    if(result != 0)
    {
        throw 1;
    }
}
}

__global__ void identity(size_t n, unsigned char* src, unsigned char* dest)
{
    printf("a");
    int index = threadIdx.x;
    int stride = blockDim.x;
    for(int i = index; i < n; i += stride)
    {
        dest[i] = src[i];
    }

    /*
    for(size_t i = 0; i < n; i++)
    {
        dest[i] = src[i];
    }
    */
}

int main(int argc, char* argv[])
{
    std::cout << "Bob's cuda testbed\n\n";

    std::chrono::high_resolution_clock::time_point startTime(
        std::chrono::high_resolution_clock::now());

    try
    {
        std::ofstream ostr(logFile, std::ios::trunc);
        ostr.close();

        size_t nBytes(2000000000);

        if(argc > 1)
        {
            nBytes = atol(argv[1]);
        }

        std::cout << format(nBytes) << " bytes\n\n";

        LOG(format(nBytes) << " bytes\n\n");

        unsigned char* src;
        unsigned char* dest;

        hipError_t rc(hipSuccess);

        // Allocate Unified Memory � accessible from CPU or GPU
        rc = hipMallocManaged(&src, nBytes);
        writeResult(rc, "hipMallocManaged");

        rc = hipMallocManaged(&dest, nBytes);
        writeResult(rc, "hipMallocManaged");

        // initialize x and y arrays on the host
        for(size_t i(0); i < nBytes; ++i)
        {
            src[i] = i % 256;
            dest[i] = 0;
        }

        int const numBlocks(1);
        int const numThreadsPerThreadBlock(256);

        LOG("Start call kernel\n");
        identity<<<numBlocks, numThreadsPerThreadBlock>>>(nBytes, src, dest);
        LOG("Finish call kernel\n");

        LOG("Start synchronize\n");
        hipDeviceSynchronize();
        LOG("Finish synchronize\n");

        std::cout << "Checking result\n";

        for(size_t i(0); i < nBytes; ++i)
        {
            if(src[i] != dest[i])
            {
                std::cout << "Match failed at byte # " << i << "\n";
            }
        }

        std::cout << "Finished checking result\n";
    }
    catch(...)
    {
        std::cout << "Caught exception\n";
    }

    std::cout << std::fixed << std::setprecision(9);
    std::cout << std::chrono::duration_cast<std::chrono::duration<double>>(
                     std::chrono::high_resolution_clock::now() - startTime)
                     .count();
    std::cout << "\n";

    return 0;
}
